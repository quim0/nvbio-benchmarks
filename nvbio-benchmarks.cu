#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021 Quim Aguado
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy of
 * this software and associated documentation files (the "Software"), to deal in
 * the Software without restriction, including without limitation the rights to
 * use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of
 * the Software, and to permit persons to whom the Software is furnished to do so,
 * subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS
 * FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR
 * COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER
 * IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
 * CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */

#include <fstream>
#include <iostream>
#include <cstdlib>
#include <cstring>
#include <chrono>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <nvbio/basic/types.h>
#include <nvbio/basic/vector.h>
#include <nvbio/alignment/alignment.h>
#include <nvbio/alignment/alignment_base.h>
#include <nvbio/alignment/batched.h>

#define TIMER_INIT  std::chrono::steady_clock::time_point bm_timer_begin; \
                    std::chrono::steady_clock::time_point bm_timer_end;
#define TIMER_START bm_timer_begin = std::chrono::steady_clock::now();
#define TIMER_STOP  bm_timer_end = std::chrono::steady_clock::now();

#define TIMER_MS std::chrono::duration_cast<std::chrono::milliseconds> (bm_timer_end - bm_timer_begin).count()

#define ALPHABET_SIZE 4

const uint32_t BAND_LEN = 31;

const char *USAGE_STR = "Usage:\n"
                        "nvbio-benchmark <file> <max_seq_len> <num_alignments> "
                        "<batch_size=50000>";


class Sequences {
public:
    size_t seq_len;
    size_t num_alignments;
    char* sequences_buffer;
    int* sequences_len;

    Sequences (char* filepath, int seq_len, int num_alignments) :\
                                                    seq_len(seq_len),
                                                    num_alignments(num_alignments) {
        std::cout << "Sequences object:" << std::endl
                  << "\tFile: " << filepath << std::endl
                  << "\tSequence length: " << seq_len << std::endl
                  << "\tNumber of alignments: " << num_alignments << std::endl;

        std::size_t seq_bytes_to_alloc = ((size_t)num_alignments * (size_t)seq_len * 2L);
        std::cout << "Allocating " << (seq_bytes_to_alloc / (1 << 20))
                  << "MiB of memory to store the sequences" << std::endl;
        try {
            this->sequences_buffer = new char[seq_bytes_to_alloc];
        } catch (std::bad_alloc & exception) {
            std::cerr << "bad_alloc detected: " << exception.what();
            exit(-1);
        }
        memset(this->sequences_buffer, 0, seq_bytes_to_alloc);
        this->sequences_len = new int[(size_t)num_alignments * 2L];

        std::ifstream file(filepath, std::ios::binary | std::ios::ate);
        if (file.fail()) {
            std::cerr << "Could not open file: \"" << filepath << "\"" << std::endl;
            // TODO
            exit(-1);
        }

        std::streamsize size = file.tellg();
        file.seekg(0, std::ios::beg);

        TIMER_INIT

        TIMER_START

        std::string line;
        size_t sequences_read = 0;
        while(std::getline(file, line) && sequences_read < (num_alignments*2)) {
            strncpy(this->get_sequence(sequences_read),
                    // +1 to avoid the initial > and <
                    line.c_str() + 1,
                    seq_len);
            this->sequences_len[sequences_read] = line.length() - 1;
            sequences_read++;
        }

        TIMER_STOP
        std::cout << "Read " << sequences_read << " sequences in " << TIMER_MS
                  << "ms." << std::endl;
    };

    ~Sequences () {
        delete [] this->sequences_buffer;
        delete [] this->sequences_len;
    }

    char* get_sequence(size_t n) const {
#ifdef DEBUG
        // Only for debug purposes
        if (n >= this->num_alignments*2) {
            std::cout << "Trying to read too far... n=" << n << std::endl;
            return 0;
        }
#endif
        return this->sequences_buffer + (this->seq_len * n);
    }
};

// Function based on the example in the NVidia blog, but adapted:
//     https://developer.nvidia.com/blog/accelerating-bioinformatics-nvbio/
// Time is returned in ms
void batch_alignment_test (const Sequences &sequences, const size_t batch_offset,
                           const uint32_t batch_size, double *time) {
    using namespace nvbio;
    // build two concatenated string sets, one for the patterns, 
    // containing a concatenated sequence of strings of 100 
    // characters each, and one for the texts, 
    // containing 200 characters each
    const uint32 n_strings   = batch_size;
    const uint32 pattern_len = sequences.seq_len;
    const uint32 text_len    = sequences.seq_len;

#ifdef DEBUG
    std::cerr << "Batch size: " << n_strings << std::endl
              << "Pattern length: " << pattern_len << std::endl
              << "Text length: " << text_len << std::endl;
#endif

    // setup the strings on the host
    nvbio::vector<host_tag, uint8> h_pattern(n_strings * pattern_len);
    nvbio::vector<host_tag, uint8> h_text(n_strings * text_len);

    // Copy patterns for this batch
    for (uint32 i = 0; i < n_strings; i++) {
        //std::cout << "copying pattern "<< i << std::endl;
        memcpy((void*)&h_pattern[i * pattern_len], 
               sequences.get_sequence(batch_offset + i*2),
               sequences.seq_len);
    }
    // Copy texts for this batch
    for (uint32 i = 0; i < n_strings; i++) {
        memcpy((void*)&h_text[i * text_len], 
               sequences.get_sequence(batch_offset + (i+1)*2),
               sequences.seq_len);
    }

    TIMER_INIT

    TIMER_START
    // copy the strings storage to the device
    nvbio::vector<device_tag, uint8> d_pattern( h_pattern );
    nvbio::vector<device_tag, uint8> d_text( h_text );

    // allocate two vectors representing the string offsets
    nvbio::vector<device_tag, uint32> d_pattern_offsets( n_strings+1 );
    nvbio::vector<device_tag, uint32> d_text_offsets( n_strings+1 );

    // prepare the string offsets using Thrust's sequence() 
    // function, setting up the offset of pattern i as i * pattern_len, 
    // and the offset of text i as i * text_len
    thrust::sequence( d_pattern_offsets.begin(), 
                      d_pattern_offsets.end(), 0u, pattern_len );
    thrust::sequence( d_text_offsets.begin(), 
                      d_text_offsets.end(), 0u, text_len );

    // prepare a vector of alignment sinks
    nvbio::vector<device_tag, aln::BestSink<uint32> > 
        sinks( n_strings );

    // and execute the batch alignment, on a GPU device
    aln::batch_banded_alignment_score<BAND_LEN>(
        aln::make_edit_distance_aligner
            <aln::GLOBAL, aln::MyersTag<ALPHABET_SIZE> >(),
        make_concatenated_string_set( n_strings, 
                                      d_pattern.begin(),
                                      d_pattern_offsets.begin() ),
        make_concatenated_string_set( n_strings, 
                                      d_text.begin(), 
                                      d_text_offsets.begin() ),
        sinks.begin(),
        aln::DeviceThreadScheduler(),
        sequences.seq_len,
        sequences.seq_len );
    TIMER_STOP
    *time += TIMER_MS;
}

int main (int argc, char* argv[]) {
    char* filepath;
    size_t batch_size = 50000;
    int seq_size = 0;
    int num_alignments;

    if (argc >= 4) {
        filepath = argv[1];
        seq_size = std::atoi(argv[2]);
        num_alignments = std::atoi(argv[3]);
    }
    if (argc == 5) {
        batch_size = std::atoi(argv[4]);
    }
    if (argc < 4 || argc > 5) {
        std::cerr << USAGE_STR << std::endl;
        return EXIT_FAILURE;
    }

    if (batch_size > num_alignments) {
        std::cerr << "Batch size can not be bigger than the number of alignments"
                  << "\nChanging batch size to " << num_alignments << std::endl;
        batch_size = num_alignments;
    } else {
        std::cout << "Batch size set to " << batch_size << std::endl;
    }

    Sequences sequences(filepath, seq_size, num_alignments);

    // Total time in milliseconds
    double total_time = 0;
    size_t alignments_computed = 0;
    int cnt = 0;

    while (alignments_computed < num_alignments) {
        size_t curr_batch_size = std::min(batch_size,
                                          num_alignments - alignments_computed);

        batch_alignment_test (sequences,
                              alignments_computed,
                              curr_batch_size,
                              &total_time);
        std::cerr << "Batch " << cnt++ << " executed." << std::endl;
        alignments_computed += batch_size;
    }

    std::cout << "Executed " << num_alignments << " alignments in "
              << total_time << "ms."
              << std::endl
              << "Performance: "
              << (double)((num_alignments * (uint64_t)(seq_size*seq_size)) / (total_time/1000)) / 1000000000
              << " GCUPs" << std::endl;

    return 0;
}
